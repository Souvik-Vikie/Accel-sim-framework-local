#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void simple_add(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 256;
    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
        h_c[i] = 0;   // Initialize host result array
    }

    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_c, 0, N * sizeof(int));  // Initialize device result array

    // <<<blocks, threads>>> : one block of 256 threads
    simple_add<<<1, 256>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize(); // Wait for kernel to finish

    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
